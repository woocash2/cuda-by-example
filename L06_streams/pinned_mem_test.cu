#include <iostream>
#include <hip/hip_runtime.h>
#include "cuda_errchk.h"

using namespace std;


const int SIZE = 1024 * 1024 * 100; // 100MB


float mallocTest(bool toDevice) {
    hipEvent_t start, stop;
    errchk(hipEventCreate(&start));
    errchk(hipEventCreate(&stop));
    errchk(hipEventRecord(start, 0));

    unsigned char * a = (unsigned char *) malloc(SIZE);
    unsigned char * devA;
    errchk(hipMalloc(&devA, SIZE));


    for (int i = 0; i < 100; i++) {
        if (toDevice)
            errchk(hipMemcpy(devA, a, SIZE, hipMemcpyHostToDevice));
        else
            errchk(hipMemcpy(a, devA, SIZE, hipMemcpyDeviceToHost));
    }

    free(a);
    hipFree(devA);

    errchk(hipEventRecord(stop, 0));
    errchk(hipEventSynchronize(stop));
    float elapsedTime;
    errchk(hipEventElapsedTime(&elapsedTime, start, stop));

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsedTime;
}


float cudaHostAallocTest(bool toDevice) {
    hipEvent_t start, stop;
    errchk(hipEventCreate(&start));
    errchk(hipEventCreate(&stop));
    errchk(hipEventRecord(start, 0));

    unsigned char * a;
    errchk(hipHostAlloc(&a, SIZE, hipHostMallocDefault));
    unsigned char * devA;
    errchk(hipMalloc(&devA, SIZE));


    for (int i = 0; i < 100; i++) {
        if (toDevice)
            errchk(hipMemcpy(devA, a, SIZE, hipMemcpyHostToDevice));
        else
            errchk(hipMemcpy(a, devA, SIZE, hipMemcpyDeviceToHost));
    }

    hipHostFree(a);
    hipFree(devA);

    errchk(hipEventRecord(stop, 0));
    errchk(hipEventSynchronize(stop));
    float elapsedTime;
    errchk(hipEventElapsedTime(&elapsedTime, start, stop));

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsedTime;
}


int main() {

    cout << "GPU times of copying 100MB:" << endl;
    cout << "C malloc host to device: " << mallocTest(true) << "ms" << endl;
    cout << "C malloc device to host: " << mallocTest(false) << "ms" << endl;
    cout << "hipHostAlloc host to device: " << cudaHostAallocTest(true) << "ms" << endl;
    cout << "hipHostAlloc device to host: " << cudaHostAallocTest(false) << "ms" << endl;


    return 0;
}