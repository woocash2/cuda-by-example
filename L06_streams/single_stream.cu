#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "cuda_errchk.h"

using namespace std;

const int N = 1024 * 1024;
const int SIZE = N * 20;


__global__ void avg3BlockCUDA(int * a, int * b, int * c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        float as = (a[idx] + a[(idx + 1) % 256] + a[(idx + 2) % 256]) / 3.0f;
        float bs = (b[idx] + b[(idx + 1) % 256] + b[(idx + 2) % 256]) / 3.0f;
        c[idx] = (as + bs) / 2.0f;
    }
}


int main() {
     // init events
    hipEvent_t start, stop;
    errchk(hipEventCreate(&start));
    errchk(hipEventCreate(&stop));
    errchk(hipEventRecord(start, 0));

    // Get our device properties
    hipDeviceProp_t prop;
    int devIdx;
    errchk(hipGetDevice(&devIdx));
    errchk(hipGetDeviceProperties(&prop, devIdx));

    // Check if out device handles overlaps
    // (Ability to perform kernels and memory copying host <-> device simultaneously)
    if (!prop.deviceOverlap) {
        cout << "No speedup from streams\n";
        return 0;
    }

    // creating a stream
    hipStream_t stream;
    errchk(hipStreamCreate(&stream));

    // host and device buffers
    int * a, * b, * c;
    int * devA, * devB, * devC;

    // allocating dev buffers
    errchk(hipMalloc(&devA, N * sizeof(int)));
    errchk(hipMalloc(&devB, N * sizeof(int)));
    errchk(hipMalloc(&devC, N * sizeof(int)));

    // allocating host buffers. We need hipHostAlloc because
    // cudaStream requires page-locked memory
    errchk(hipHostAlloc(&a, SIZE * sizeof(int), hipHostMallocDefault));
    errchk(hipHostAlloc(&b, SIZE * sizeof(int), hipHostMallocDefault));
    errchk(hipHostAlloc(&c, SIZE * sizeof(int), hipHostMallocDefault));

    // filling host buffers with random values
    for (int i = 0; i < SIZE; i++) {
        a[i] = rand() % 1000000;
        b[i] = rand() % 1000000;
    }

    // call kernels by chunks
    for (int i = 0; i < SIZE; i += N) {
        // Copy chunks of size N. We use hipMemcpyAsync as it will turn out being important
        // when we will use more than one stream
        errchk(hipMemcpyAsync(devA, a + i, N * sizeof(int), hipMemcpyHostToDevice, stream));
        errchk(hipMemcpyAsync(devB, b + i, N * sizeof(int), hipMemcpyHostToDevice, stream));

        // kernel call
        avg3BlockCUDA<<<N / 256, 256, 0, stream>>>(devA, devB, devC);

        // place the chunk back to the host
        errchk(hipMemcpyAsync(c + i, devC, N * sizeof(int), hipMemcpyDeviceToHost, stream));
    }

    // Because our memcpys and kernel calls were async, we need to make sure
    // that everything is finished before freeing the memory
    errchk(hipStreamSynchronize(stream));

    // free dev memory
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);

    // free host memory using hipHostFree
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);

    // Destroy stream
    hipStreamDestroy(stream);

    // measure time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "GPU time: " << elapsedTime << "ms\n";

    // destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
