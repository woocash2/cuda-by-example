#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "cuda_errchk.h"

using namespace std;

const int N = 1024 * 1024;
const int SIZE = N * 20;


__global__ void avg3BlockCUDA(int * a, int * b, int * c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        float as = (a[idx] + a[(idx + 1) % 256] + a[(idx + 2) % 256]) / 3.0f;
        float bs = (b[idx] + b[(idx + 1) % 256] + b[(idx + 2) % 256]) / 3.0f;
        c[idx] = (as + bs) / 2.0f;
    }
}


int main() {
     // init events
    hipEvent_t start, stop;
    errchk(hipEventCreate(&start));
    errchk(hipEventCreate(&stop));
    errchk(hipEventRecord(start, 0));

    // Get our device properties
    hipDeviceProp_t prop;
    int devIdx;
    errchk(hipGetDevice(&devIdx));
    errchk(hipGetDeviceProperties(&prop, devIdx));

    // Check if out device handles overlaps
    // (Ability to perform kernels and memory copying host <-> device simultaneously)
    if (!prop.deviceOverlap) {
        cout << "No speedup from streams\n";
        return 0;
    }

    // creating a stream
    hipStream_t stream[2];
    errchk(hipStreamCreate(&stream[0]));
    errchk(hipStreamCreate(&stream[1]));

    // host and device buffers
    int * a, * b, * c;
    int * devA[2], * devB[2], * devC[2];

    // allocating dev buffers
    for (int i = 0; i < 2; i++) {
        errchk(hipMalloc(&devA[i], N * sizeof(int)));
        errchk(hipMalloc(&devB[i], N * sizeof(int)));
        errchk(hipMalloc(&devC[i], N * sizeof(int)));
    }

    // allocating host buffers. We need hipHostAlloc because
    // cudaStream requires page-locked memory
    errchk(hipHostAlloc(&a, SIZE * sizeof(int), hipHostMallocDefault));
    errchk(hipHostAlloc(&b, SIZE * sizeof(int), hipHostMallocDefault));
    errchk(hipHostAlloc(&c, SIZE * sizeof(int), hipHostMallocDefault));

    // filling host buffers with random values
    for (int i = 0; i < SIZE; i++) {
        a[i] = rand() % 1000000;
        b[i] = rand() % 1000000;
    }

    // call kernels by chunks
    for (int i = 0; i < SIZE; i += 2 * N) {
        for (int j = 0; j < 2; j++) {
            // Copy chunks of size N. We use hipMemcpyAsync as it will turn out being important
            // when we will use more than one stream
            errchk(hipMemcpyAsync(devA[j], a + i + j * N, N * sizeof(int), hipMemcpyHostToDevice, stream[j]));
            errchk(hipMemcpyAsync(devB[j], b + i + j * N, N * sizeof(int), hipMemcpyHostToDevice, stream[j]));

            // kernel call
            avg3BlockCUDA<<<N / 256, 256, 0, stream[j]>>>(devA[j], devB[j], devC[j]);

            // place the chunk back to the host
            errchk(hipMemcpyAsync(c + i + j * N, devC[j], N * sizeof(int), hipMemcpyDeviceToHost, stream[j]));
        }
    }

    // Because our memcpys and kernel calls were async, we need to make sure
    // that everything is finished before freeing the memory
    for (int i = 0; i < 2; i++) {
        errchk(hipStreamSynchronize(stream[i]));
    }

    // free dev memory
    for (int i = 0; i < 2; i++) {
        hipFree(devA[i]);
        hipFree(devB[i]);
        hipFree(devC[i]);
    }

    // free host memory using hipHostFree
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);

    // Destroy stream
    for (int i = 0; i < 2; i++)
        hipStreamDestroy(stream[i]);

    // measure time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "GPU time: " << elapsedTime << "ms\n";

    // destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
